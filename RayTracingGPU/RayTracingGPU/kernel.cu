#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include "image_cuda.h"
#include "vec3_cuda.h"
#include "ray_cuda.h"
#include "hitable_cuda.h"
#include "material_cuda.h"
#include "sphere_cuda.h"
#include "world_cuda.h"
#include <hiprand/hiprand_kernel.h>
#include "camera_cuda.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
#define RND (hiprand_uniform(&local_rand_state))

using namespace hittable;

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 get_color(const ray& r, hittable::world& world, hiprandState* local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = make_vec3(1.0f, 1.0f, 1.0f);
   
    // 50 iterations for ray bounce
    for (int i = 0; i < 50; i++) {
        hittable::record rec;

        if (hit(world, cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (scatter(*rec.material, cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return make_vec3(0.0f, 0.0f, 0.0f);
            }
        }
        else {
            vec3 unit_direction = make_unit(cur_ray.direction);
            float t = 0.5f * (unit_direction.y + 1.0f);
            vec3 c = (1.0f - t) * make_vec3(1.0f, 1.0f, 1.0f) + t * make_vec3(0.5f, 0.7f, 1.0f);
            return cur_attenuation * c;
        }
    }
}

__global__ void rand_world_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void rand_pixels_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprand_init(2024 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(
    vec3* image, 
    int max_x, 
    int max_y, 
    int ns, 
    camera* cam, 
    hittable::world* world, 
    hiprandState* rand_state
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    
    color col = make_vec3(0.f, 0.f, 0.f);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = get_ray(*cam, u, v, &local_rand_state);
        col += get_color(r, *world, &local_rand_state);
    }

    rand_state[pixel_index] = local_rand_state;
    
    col /= float(ns);
    image[pixel_index] = col;
}

__global__ void create_world(hittable::world* world, camera* camera, int nx, int ny, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;

        sphere d_list[22 * 22 + 1 + 3];
        d_list[0] = make_sphere(make_vec3(0.f, -1000.0f, -1.f), 1000.f, &make_lambertian(make_vec3(0.5f, 0.5f, 0.5f)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center = make_vec3(a + RND, 0.2f, b + RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = make_sphere(center, 0.2f,
                        &make_lambertian(make_vec3(RND * RND, RND * RND, RND * RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = make_sphere(center, 0.2f,
                        &make_metal(make_vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
                }
                else {
                    d_list[i++] = make_sphere(center, 0.2f, &make_dielectric(1.5f));
                }
            }
        }

        d_list[i++] = make_sphere(make_vec3(0.f, 1.f, 0.f), 1.0f, &make_dielectric(1.5f));
        d_list[i++] = make_sphere(make_vec3(-4.f, 1.f, 0.f), 1.0f, &make_lambertian(make_vec3(0.4f, 0.2f, 0.1f)));
        d_list[i++] = make_sphere(make_vec3(4.f, 1.f, 0.f), 1.0f, &make_metal(make_vec3(0.7f, 0.6f, 0.5f), 0.0f));
        world = &make_world(d_list, 22 * 22 + 1 + 3);

        *rand_state = local_rand_state;

        vec3 lookfrom = make_vec3(13.f, 2.f, 3.f);
        vec3 lookat = make_vec3(0.f, 0.f, 0.f);
        float dist_to_focus = length(lookfrom - lookat);
        float aperture = 0.1;
        camera = &make_camera(
            lookfrom,
            lookat,
            make_vec3(0.f, 1.f, 0.f),
            30.0f,
            float(nx) / float(ny),
            aperture,
            dist_to_focus
        );
    }
}


__global__ void free_world(world* d_world, camera* d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete (d_world->objects[i]).material;
    }
    delete [] d_world->objects;
    delete d_world;
    delete d_camera;
}

int main() {
    int nx = 800; // width
    int ny = 450; // heigth
    int ns = 500; // numar de sample uri
    int tx = 8;
    int ty = 8;

    Image image(nx, ny);

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t pixels_size = num_pixels * sizeof(color);
    color* image_pixels;
    checkCudaErrors(hipMallocManaged((void**)&image_pixels, pixels_size));
    
    // allocate random state
    hiprandState* d_rand_state_pixels;
    checkCudaErrors(hipMalloc((void**)&d_rand_state_pixels, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state_world;
    checkCudaErrors(hipMalloc((void**)&d_rand_state_world, 1 * sizeof(hiprandState)));

    //world
    world* d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(world*)));

    // camera
    camera* d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera)));

    //// clock to measure time
    clock_t start, stop;
    start = clock();

    // we need that 2nd random state to be initialized for the world creation
    rand_world_init <<< 1, 1 >>> (d_rand_state_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // create world
    create_world <<<1, 1 >>> (d_world, d_camera, nx, ny, d_rand_state_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // calculate blocks and threads
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    // init random state for each pixel
    rand_pixels_init <<< blocks, threads >> > (nx, ny, d_rand_state_pixels);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // render the scene
    render <<< blocks, threads >>> (image_pixels, nx, ny, ns, d_camera, d_world, d_rand_state_pixels);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";



    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            image.setPixel(ny - 1 - j, i, image_pixels[pixel_index]);
        }
    }

    image.displayImage();

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world <<<1, 1 >>> (d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_rand_state_pixels));
    checkCudaErrors(hipFree(d_rand_state_world));
    checkCudaErrors(hipFree(image_pixels));
    hipDeviceReset();
}
